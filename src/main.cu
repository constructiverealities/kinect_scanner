#include "ncurses_3dscanner.hpp"

int main(int argc, char *argv[]) {
	NCurses_3dscanner ncurse;
	char cmd;
	noecho();
	do{
		timeout(10);
		cmd = mvgetch(4,0);
		switch (cmd){
			case '0':
				ncurse.initialize();
				break;
			case '1':
				ncurse.scan();
				break;
            case '2':
                ncurse.saveMesh();
                break;
		}
	}while( cmd != '9');
	return 0;
//	if(scanner.intializeGridPosition()){
//		scanner.scan();
//		scanner.extractMesh();
//		scanner.saveMesh();
//		return 0;
//	}
//	std::cout << "could not initialize grid location" << std::endl;
//	return 1;
}
